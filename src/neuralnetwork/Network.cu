#include "../includes.h"
#include "Network.h"

__constant__ int zero = 0;
int* zero_pointer;
float* f_zero_pointer;

std::ostream& operator<<(std::ostream& os, const Network& network) {
    os << "Network with " << network.L << " layers:\n";
    for (int i = 0; i < network.L; i++) {
        float* weights = NULL;
        hipMemcpy(weights,  network.layers[i]->dev_weights, network.layers[i]->weights_size*sizeof(float), hipMemcpyDeviceToHost);
        float* biases = NULL;
        hipMemcpy(biases,  network.layers[i]->dev_biases, network.layers[i]->biases_size*sizeof(float), hipMemcpyDeviceToHost);

        os << "\tLayer " << i << ":\n";
        os << "\t\tWeights of size " << network.layers[i]->weights_size <<":\n\t\t";
        for (int j = 0; j < network.layers[i]->weights_size; j++) {
            os << weights[j] << ", ";
        }
        os << "\b\b\n\t\tBiases:\n\t\t";
        for (int j = 0; j < network.layers[i]->biases_size; j++) {
            os << biases[j] << ", ";
        }
        os << "\b\b\n";

        delete [] weights;
        delete [] biases;
    }
    os << "\n\n";
    return os;
}

Network::Network() {
    hipGetSymbolAddress((void**) &zero_pointer, zero);
    hipGetSymbolAddress((void**) &f_zero_pointer, zero);
}

void Network::init(layer_data* layers, int L, hyperparams params) {

    this->L = L;
    this->params = params;
    hipMalloc((void**) &dev_params, sizeof(hyperparams));
    hipMemcpy(dev_params, &params, sizeof(hyperparams), hipMemcpyHostToDevice);
    this->layers = new std::unique_ptr<layer>[L];

    // initialize layers
    for (int l = 0; l < L; l++) {
        std::unique_ptr<layer> new_layer = nullptr;
        switch (layers[l].type) {
            case LAYER_NUM_INPUT:
                new_layer = std::make_unique<input_layer>();
                break;
            case LAYER_NUM_CONVOLUTIONAL:
                new_layer = std::make_unique<convolutional_layer>();
                break;
            case LAYER_NUM_FULLY_CONNECTED:
                new_layer = std::make_unique<fully_connected_layer>();
                break;
        }
        layer_data previous_data;
        float* new_delta;
        if (l > 0) {
            previous_data = this->layers[l - 1]->data;
            new_delta = this->layers[l-1]->delta;
        }
        new_layer->init(layers[l], previous_data, new_delta);
        this->layers[l] = move(new_layer);
    }

    int elems = this->layers[L-1]->data.elems+OUTPUT_NEURONS;

    hipMalloc((void**) &activations, elems*sizeof(float));
    hipMalloc((void**) &derivatives_z, elems*sizeof(float));
}

void Network::feedforward(float* a, float* dev_activations, float* dev_derivatives_z) {
    hipMemcpy(dev_activations, a, INPUT_NEURONS*sizeof(float), hipMemcpyDeviceToDevice);

    for (int l = 1; l < L; l++) {
        layers[l]->feedforward(dev_activations, dev_derivatives_z);
    }
}

std::pair<int,int> Network::evaluate(std::vector<std::pair<float*,float*>> test_data, int test_data_size) {
    auto start = std::chrono::high_resolution_clock::now();

    int* dev_correct;
    hipMalloc((void**) &dev_correct, sizeof(int));
    hipMemcpy(dev_correct, zero_pointer, sizeof(int), hipMemcpyDeviceToDevice);

    for (int k = 0; k < (int) test_data_size; k++) {
        feedforward(test_data[k].first, activations, derivatives_z);
        hipDeviceSynchronize();
        eval<<<1,1>>>(test_data[k].second, &activations[layers[L-1]->data.elems], dev_correct, &layers[L-1]->dev_data->n_out.x);
    }
    hipDeviceSynchronize();

    int correct;
    hipMemcpy(&correct, dev_correct, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_correct);
    auto end = std::chrono::high_resolution_clock::now();
    return {correct, std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()};
}

void Network::SGD(std::vector<std::pair<float*,float*>> training_data, std::vector<std::pair<float*,float*>> test_data) {

    auto ev = evaluate(test_data, params.test_data_size);
    auto correct = ev.first;
    auto durationEvaluate = ev.second;

    if (params.test_data_size > 0) {
        std::cerr << "0 Accuracy: " << (float) correct / params.test_data_size << " evaluated in " << durationEvaluate << "ms\n";
    }

    for (int i = 0; i < params.epochs; i++) {
        // time the epoch
        auto start = std::chrono::high_resolution_clock::now();

        //std::cerr << i+1 << " ";

        // obtain a time-based seed
        unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
        shuffle(training_data.begin(), training_data.end(), std::default_random_engine(seed));

        // create mini batches and update them
        std::vector<std::pair<float*,float*>> mini_batch (params.mini_batch_size, {nullptr, nullptr});
        for (int j = 0; j < params.training_data_size / params.mini_batch_size; j++) {
            for (int k = 0; k < params.mini_batch_size; k++) {
                mini_batch[k].first = training_data[j * params.mini_batch_size + k].first;
                mini_batch[k].second = training_data[j * params.mini_batch_size + k].second;
            }
            update_mini_batch(mini_batch);
        }

        // end the timer
        auto end = std::chrono::high_resolution_clock::now();
        auto durationTrain = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

        // evaluate the network
        ev = evaluate(test_data, params.test_data_size);
        correct = ev.first;
        durationEvaluate = ev.second;

       if (params.test_data_size > 0)
           std::cerr << "Accuracy: " << (float) correct / params.test_data_size << ", trained in " << durationTrain << "ms, evaluated in " << durationEvaluate << "ms\n";

        // reduce learning rate
	    if (i < 100) {
            params.fully_connected_biases_learning_rate -= params.fcBRed;
            params.fully_connected_weights_learning_rate -= params.fcWRed;
            params.convolutional_biases_learning_rate -= params.convBRed;
            params.convolutional_weights_learning_rate -= params.convWRed;
            hipMemcpy(dev_params, &params, sizeof(hyperparams), hipMemcpyHostToDevice);
        }
    }
}

void Network::update_mini_batch(std::vector<std::pair<float*,float*>> mini_batch) {

    for (int num = 0; num < params.mini_batch_size; num++) {
        backprop(mini_batch[num].first, mini_batch[num].second);
    }

    // update velocities
    for (int i = 1; i < L; i++) layers[i]->update(dev_params);
}

void Network::backprop(float* in, float* out) {
    // feedfoward
    feedforward(in, activations, derivatives_z);

    // backpropagate
    set_delta<<<OUTPUT_NEURONS,1>>> (layers[L-1]->delta, &activations[layers[L-1]->data.elems], out, &dev_params->cost);

    for (int l = L - 1; l >= 1; l--) {
        layers[l]->backprop(activations, derivatives_z);
    }
}

void Network::save(std::string filename) {
    std::ofstream file(filename);
    file << L << "\n";
    file.close();

    for (int l = 0; l < L; l++) layers[l]->save(filename);
}

void Network::load(std::string filename, hyperparams params, layer_data* &layers) {
    std::ifstream file;

    file.open(filename);
    std::string line;
    std::string str;
    getline(file, line);
    L = atoi(line.c_str());

    layers = new layer_data[L];
    float** biases = new float* [L];
    float** biases_vel = new float* [L];
    float** weights = new float* [L];
    float** weights_vel = new float* [L];

    for (int l = 0; l < L; l++) {
        getline(file, line); // get line
        std::stringstream ss(line);
        getline(ss, str, DEL); // get first string before DEL
        std::unique_ptr<layer> new_layer = nullptr;

        int type = atoi(str.c_str());
        layers[l].type = type;

        switch (layers[l].type) {
            case LAYER_NUM_INPUT:
                new_layer = std::make_unique<input_layer>();
                break;
            case LAYER_NUM_CONVOLUTIONAL:
                new_layer = std::make_unique<convolutional_layer>();
                break;
            case LAYER_NUM_FULLY_CONNECTED:
                new_layer = std::make_unique<fully_connected_layer>();
                break;
        }

        new_layer->load(line, &layers[l], biases[l], biases_vel[l], weights[l], weights_vel[l]);

    }

    init(layers, L, params);

    for (int l = 1; l < L; l++) {
        // copy the loaded weights to layer
        hipMemcpy(this->layers[l]->dev_biases, biases[l], this->layers[l]->biases_size*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(this->layers[l]->dev_biases_vel, biases_vel[l], this->layers[l]->biases_size*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(this->layers[l]->dev_weights, weights[l], this->layers[l]->weights_size*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(this->layers[l]->dev_weights_vel, weights_vel[l], this->layers[l]->weights_size*sizeof(float), hipMemcpyHostToDevice);
    }

    // free memory
    file.close();
    for (int l = 1; l < L; l++) {
        delete[] biases[l];
        delete[] biases_vel[l];
        delete[] weights[l];
        delete[] weights_vel[l];
    }
    delete[] biases;
    delete[] biases_vel;
    delete[] weights;
    delete[] weights_vel;
}

void Network::clear() {
    for (int l = 0; l < L; l++) layers[l]->clear();

    hipFree(dev_params);
    hipFree(activations);
    hipFree(derivatives_z);
    delete[] layers;
}